#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <chrono>

std::chrono::duration<double> read_time(0);
std::chrono::duration<double> l_time(0);
std::chrono::duration<double> u_time(0);
std::chrono::duration<double> lu_decomposition_time(0);
std::chrono::duration<double> total_time(0);

#define TILE 100

void readInput(const char *filename, int &N, double **A, double **B) {
    std::ifstream infile(filename);
    if (!infile) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    infile >> N;

    *A = (double *)malloc(N * N * sizeof(double));
    *B = (double *)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            infile >> (*A)[i * N + j];

    for (int i = 0; i < N; ++i)
        infile >> (*B)[i];

    infile.close();
}

__global__ void luDecomposition(double *A, double *L, double *U, int N) {
    int j = threadIdx.x;
    int i = blockIdx.x;
    int row = i * TILE + j;
    printf("Thread (%d, %d) - Row: %d\n", i, j, row);
    // auto start = std::chrono::high_resolution_clock::now();
    // if (row < N) {
    //     for (int j = 0; j < N; j++) {
    //         if (j < row) {
    //             L[row * N + j] = A[row * N + j]; // L below diagonal
    //             U[row * N + j] = A[row * N + j]; // U below diagonal
    //         } else {
    //             U[row * N + j] = A[row * N + j]; // U above diagonal
    //             L[row * N + j] = (row == j) ? 1.0 : 0.0; // L diagonal elements
    //         }
    //     }
    // }
    // auto end = std::chrono::high_resolution_clock::now();
    // l_time += end - start;
    // u_time += end - start;

    // __syncthreads(); // Ensure all threads have updated L and U

    // Perform elimination
    for (int k = 0; k < N; k++) {
        if (row > k) {
            // start = std::chrono::high_resolution_clock::now();
            double factor = U[row * N + k] / U[k * N + k];
            U[row * N + k] = 0.0;
            L[row * N + k] = factor;
            // end = std::chrono::high_resolution_clock::now();
            // l_time += end - start;
            for (int j = k + 1; j < N; j++) {
                U[row * N + j] -= factor * U[k * N + j];
            }
            // end = std::chrono::high_resolution_clock::now();
            // u_time += end - start;
        }
    }
}


void forwardSubstitution(double* A, double* B, double* Y, int N) {
    for (int row = 0; row < N; row++) {
        Y[row] = B[row];
        for (int j = 0; j < row; j++) {
            Y[row] -= A[row * N + j] * Y[j];
        }
    }
}

void backwardSubstitution(double* A, double* B, double* X, int N) {
    for (int row = N - 1; row >= 0; row--) {
        X[row] = B[row];
        for (int j = row + 1; j < N; j++) {
            X[row] -= A[row * N + j] * X[j];
        }
        X[row] /= A[row * N + row];
    }
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: ./cuda " << "<input_file> " << "<output_file>" << std::endl;
        exit(EXIT_FAILURE);
    }
    int N;
    double *A, *B, *X;
    auto read_start = std::chrono::high_resolution_clock::now();
    readInput(argv[1], N, &A, &B);
    auto read_end = std::chrono::high_resolution_clock::now();
    read_time = read_end - read_start;

    X = (double *)malloc(N * sizeof(double));
    double *d_A, *d_B, *d_L, *d_U, *d_Y, *d_X;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_B, N * sizeof(double));
    hipMalloc(&d_L, N * N * sizeof(double));
    hipMalloc(&d_U, N * N * sizeof(double));
    hipMalloc(&d_Y, N * sizeof(double));
    hipMalloc(&d_X, N * sizeof(double));

    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridConfig((N + TILE - 1)/TILE, 1, 1);
    dim3 blockConfig(TILE, 1, 1);

    // int sharedMemSize = N * N * sizeof(double) + N * sizeof(double) + N * sizeof(double);

    double* L = (double*)malloc(N * N * sizeof(double));
    double* U = (double*)malloc(N * N * sizeof(double));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (j < i) {
                L[i * N + j] = A[i * N + j];
                U[i * N + j] = A[i * N + j];
            } else {
                U[i * N + j] = A[i * N + j];
                L[i * N + j] = (i == j) ? 1.0 : 0.0;
            }
        }
    }

    hipMemcpy(d_L, L, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, N * N * sizeof(double), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    luDecomposition<<<gridConfig, blockConfig>>>(d_A, d_L, d_U, N);
    auto end = std::chrono::high_resolution_clock::now();

    lu_decomposition_time = end - start;


    hipMemcpy(L, d_L, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    auto start_sub = std::chrono::high_resolution_clock::now();
    forwardSubstitution(L, B, X, N);
    backwardSubstitution(U, X, X, N);
    auto end_sub = std::chrono::high_resolution_clock::now();

    total_time =  lu_decomposition_time + end_sub - start_sub;

    std::cout << "Read time: " << read_time.count() << "s" << std::endl;
    std::cout << "LU decomposition time: " << lu_decomposition_time.count() << "s" << std::endl;
    std::cout << "Total time: " << total_time.count() << "s" << std::endl;

    std::ofstream outfile(argv[2]);
    if (!outfile) {
        std::cerr << "Error opening file for writing: "<< argv[2] << std::endl;
        exit(EXIT_FAILURE);
    }

    outfile << N << std::endl;

    // Write L matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << L[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write U matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << U[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write solution vector X
    for (int i = 0; i < N; i++) {
        outfile << X[i] << std::endl;
    }

    outfile.close();

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_L);
    hipFree(d_U);
    hipFree(d_Y);
    hipFree(d_X);

    // Free host memory
    free(A);
    free(B);
    free(X);
    return 0;
}