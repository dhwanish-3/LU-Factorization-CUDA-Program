#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <chrono>

std::chrono::duration<double> read_time(0);
std::chrono::duration<double> l_time(0);
std::chrono::duration<double> u_time(0);
std::chrono::duration<double> lu_decomposition_time(0);
std::chrono::duration<double> total_time(0);

#define TILE 8

void readInput(const char *filename, int &N, double **A, double **B) {
    std::ifstream infile(filename);
    if (!infile) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    infile >> N;

    *A = (double *)malloc(N * N * sizeof(double));
    *B = (double *)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            infile >> (*A)[i * N + j];

    for (int i = 0; i < N; ++i)
        infile >> (*B)[i];

    infile.close();
}

void writeToFile(const char* filename, int N, double* L, double* U, double* X) {
    std::ofstream outfile(filename);
    if (!outfile) {
        std::cerr << "Error opening file for writing: "<< filename << std::endl;
        exit(EXIT_FAILURE);
    }

    outfile << N << std::endl;

    // Write L matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << L[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write U matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << U[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write X
    for (int i = 0; i < N; i++) {
        outfile << X[i] << std::endl;
    }

    outfile.close();
}

// Row elimination Kernel
__global__ void rowElimination(double* L, double* U, int N, int index) {
    int pivotRow = index * N;
    int currentRow = (blockDim.x * blockIdx.x + threadIdx.x) * N;
    int start = currentRow + index;
    int end = currentRow + N;

    extern __shared__ double Us[];
    __shared__ double pivot;
    pivot = L[start];
    for (int i = 0; i < N; i++) {
        Us[i] = U[pivotRow + i];
    }

    if (currentRow > pivotRow && currentRow < N * N) {
        for (int i = start + 1; i < end; ++i) {
            U[i] = U[i] - (pivot * Us[i - currentRow]);
        }
    }
}

// kernel without shared memory
// __global__ void rowElimination(double* L, double* U, int N, int index, int bsize) {
//     int pivotRow = index * N;
//     int currentRow = ((TILE * blockIdx.x) + threadIdx.x) * N;
//     int start = currentRow + index;
//     int end = currentRow + N;

//     if (currentRow > pivotRow && currentRow < N * N) {
//         for (int i = currentRow; i < start + 1; i++) {
//             U[i] = 0;
//         }
//         for (int i = start + 1; i < end; ++i) {
//             U[i] = U[i] - (L[start] * U[pivotRow + (i - currentRow)]);
//         }
//     }
// }

// kernel without shared memory
// __global__ void computeL(double* U, double *L, int N, int index) {
//     int id = index + threadIdx.x + 1;
//     int start = (index * N + index);
//     L[start] = 1; // diagonal elements of L
//     if (id < N) {
//         L[id * N + index] = (U[id * N + index] / U[start]);
//     }
// }

__global__ void computeL(double* U, double *L, int N, int index) {
    int id = index + threadIdx.x + 1;
    int start = (index * N + index);

    __shared__ double pivot;
    pivot = U[start];
    if (id < N) {
        L[id * N + index] = U[id * N + index] / pivot;
    }
}

void forwardSubstitution(double* L, double* B, double* Y, int N) {
    for (int row = 0; row < N; row++) {
        Y[row] = B[row];
        for (int j = 0; j < row; j++) {
            Y[row] -= L[row * N + j] * Y[j];
        }
    }
}

void backwardSubstitution(double* U, double* Y, double* X, int N) {
    for (int row = N - 1; row >= 0; row--) {
        X[row] = Y[row];
        for (int j = row + 1; j < N; j++) {
            X[row] -= U[row * N + j] * X[j];
        }
        X[row] /= U[row * N + row];
    }
}

int main(int argc, char** argv) {
    if (argc < 4) {
        std::cerr << "Usage: ./cuda " << "<input_file> <output_file> <timing_file>" << std::endl;
        exit(EXIT_FAILURE);
    }
    int N;
    double *A, *B;
    auto read_start = std::chrono::high_resolution_clock::now();
    readInput(argv[1], N, &A, &B);
    auto read_end = std::chrono::high_resolution_clock::now();
    read_time = read_end - read_start;

    double *d_L, *d_U;
    hipMalloc(&d_L, N * N * sizeof(double));
    hipMalloc(&d_U, N * N * sizeof(double));
    hipMemcpy(d_U, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    
    double* X = (double *)malloc(N * sizeof(double));
    double* Y = (double *)malloc(N * sizeof(double));
    double* L = (double*)malloc(N * N * sizeof(double));
    double* U = (double*)malloc(N * N * sizeof(double));
    
    hipEvent_t startLU, stopLU;
    hipEventCreate(&startLU);
    hipEventCreate(&stopLU);
    for (int i = 0; i < N; ++i) {
        hipEventRecord(startLU);
        
        computeL<<<1, N - i - 1>>>(d_U, d_L, N, i);
        
        hipEventRecord(stopLU);
        hipEventSynchronize(stopLU);
        float l1_time = 0;
        hipEventElapsedTime(&l1_time,startLU, stopLU);
        l_time += std::chrono::duration<double>(l1_time/1000);
        
        int sharedMem = N * sizeof(double);
        hipDeviceSynchronize();
        hipEventRecord(startLU);
        
        dim3 gridConfig(((N)/TILE) + ((N%TILE) ? 1 : 0), 1, 1);
        dim3 blockConfig(TILE, 1, 1);
        rowElimination<<<gridConfig, blockConfig, sharedMem>>>(d_L, d_U, N, i);

        hipEventRecord(stopLU);
        hipEventSynchronize(stopLU);
        float u1_time = 0;
        hipEventElapsedTime(&u1_time,startLU, stopLU);
        u_time += std::chrono::duration<double>(u1_time/1000);
    }

    hipMemcpy(L, d_L, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // removing unnecessary elements from L and U
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (i == j) {
                L[i * N + j] = 1;
            }
            if (i < j) {
                L[i * N + j] = 0;
            }
            if (i > j) {
                U[i * N + j] = 0;
            }
        }
    }
    
    auto start_sub = std::chrono::high_resolution_clock::now();
    forwardSubstitution(L, B, Y, N);
    backwardSubstitution(U, Y, X, N);
    auto end_sub = std::chrono::high_resolution_clock::now();

    lu_decomposition_time = l_time + u_time;
    total_time = read_time + l_time + u_time + end_sub - start_sub;

    // write timing to file
    std::ofstream timingFile(argv[3]);
    if (!timingFile) {
        std::cerr << "Error opening file for writing: " << argv[3] << std::endl;
        exit(EXIT_FAILURE);
    }
    timingFile << "Read time: " << read_time.count() << "s" << std::endl;
    timingFile << "L time: " << l_time.count() << "s" << std::endl;
    timingFile << "U time: " << u_time.count() << "s" << std::endl;
    timingFile << "LU decomposition time: " << lu_decomposition_time.count() << "s" << std::endl;
    timingFile << "Total time: " << total_time.count() << "s" << std::endl;
    timingFile.close();

    // write output to file
    writeToFile(argv[2], N, L, U, X);

    // Free device memory
    hipFree(d_L);
    hipFree(d_U);
    // Free host memory
    free(A);
    free(B);
    free(X);
    return 0;
}
