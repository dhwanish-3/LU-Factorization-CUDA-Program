#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

void readInput(int& N, double** A, double** B) {

}

__global__ void matrix_addition(int N, double* A, double* B, double* sol) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int index = row * N + col;

    sol[index] = A[index] + B[index];
}

int main() {
    int N = 2;
    double* A, *dA;
    double* B, *dB;
    double* sol, *dsol;

    A = (double*)malloc(N * N * sizeof(double));
    B = (double*)malloc(N * N * sizeof(double));
    sol = (double*)malloc(N * N * sizeof(double));
    for (int i = 0; i < N * N; i++) {
        A[i] = 1.0;
        B[i] = 2.0;
    }

    hipMalloc(&dA, N * N * sizeof(double));
    hipMalloc(&dB, N * N * sizeof(double));
    hipMalloc(&dsol, N * N * sizeof(double));

    hipMemcpy(dA, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridConfig(1, 1);
    dim3 blockConfig(N, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matrix_addition<<<gridConfig, blockConfig>>>(N, dA, dB, dsol);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(sol, dsol, N * N * sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", sol[i * N + j]);
        }
        printf("\n");
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix Addition time: %f ms\n", milliseconds);

    return 0;
}