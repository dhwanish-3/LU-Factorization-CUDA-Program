#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <chrono>

std::chrono::duration<double> read_time(0);
std::chrono::duration<double> l_time(0);
std::chrono::duration<double> u_time(0);
std::chrono::duration<double> lu_decomposition_time(0);
std::chrono::duration<double> total_time(0);

#define TILE 16

void readInput(const char *filename, int &N, double **A, double **B) {
    std::ifstream infile(filename);
    if (!infile) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    infile >> N;

    *A = (double *)malloc(N * N * sizeof(double));
    *B = (double *)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            infile >> (*A)[i * N + j];

    for (int i = 0; i < N; ++i)
        infile >> (*B)[i];

    infile.close();
}

void writeToFile(const char* filename, int N, double* L, double* U, double* X) {
    std::ofstream outfile(filename);
    if (!outfile) {
        std::cerr << "Error opening file for writing: "<< filename << std::endl;
        exit(EXIT_FAILURE);
    }

    outfile << N << std::endl;

    // Write L matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << L[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write U matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << U[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write X
    for (int i = 0; i < N; i++) {
        outfile << X[i] << std::endl;
    }

    outfile.close();
}

// Row elimination Kernel
__global__ void elimination(double* L, double* U, int n, int index, int bsize) {
    int idThread = threadIdx.x;
    int idBlock = blockIdx.x;

    int pivotRow = (index * n);
    int currentRow = (((bsize * idBlock) + idThread) * n);
    int start = currentRow + index;
    int end = currentRow + n;

    if (currentRow > pivotRow && currentRow < n * n) {
        for (int i = currentRow; i < start + 1; i++) {
            U[i] = 0;
        }
        for (int i = start + 1; i < end; ++i) {
            U[i] = U[i] - (L[start] * U[pivotRow + (i - currentRow)]);
        }
    }
}

__global__ void scaleIndex(double* U, double *L, int n, int index) {
    int id = index + threadIdx.x + 1;
    int start = (index * n + index);
    L[start] = 1; // diagonal elements of L
    // for (int i = index + 1; i < n; ++i) {
    //     L[i * n + index] = (U[i * n + index] / U[start]);
    // }
    if (id < n) {
        L[id * n + index] = (U[id * n + index] / U[start]);
    }
}

void forwardSubstitution(double* L, double* B, double* Y, int N) {
    for (int row = 0; row < N; row++) {
        Y[row] = B[row];
        for (int j = 0; j < row; j++) {
            Y[row] -= L[row * N + j] * Y[j];
        }
    }
}

void backwardSubstitution(double* U, double* Y, double* X, int N) {
    for (int row = N - 1; row >= 0; row--) {
        X[row] = Y[row];
        for (int j = row + 1; j < N; j++) {
            X[row] -= U[row * N + j] * X[j];
        }
        X[row] /= U[row * N + row];
    }
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: ./cuda " << "<input_file> " << "<output_file>" << std::endl;
        exit(EXIT_FAILURE);
    }
    int N;
    double *A, *B;
    auto read_start = std::chrono::high_resolution_clock::now();
    readInput(argv[1], N, &A, &B);
    auto read_end = std::chrono::high_resolution_clock::now();
    read_time = read_end - read_start;

    double *d_L, *d_U;
    hipMalloc(&d_L, N * N * sizeof(double));
    hipMalloc(&d_U, N * N * sizeof(double));
    hipMemcpy(d_U, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 gridConfig((N/TILE) + ((N%TILE) ? 1 : 0), 1, 1);
    dim3 blockConfig(TILE, 1, 1);
    
    double* X = (double *)malloc(N * sizeof(double));
    double* Y = (double *)malloc(N * sizeof(double));
    double* L = (double*)malloc(N * N * sizeof(double));
    double* U = (double*)malloc(N * N * sizeof(double));

    hipEvent_t startLU, stopLU;
    hipEventCreate(&startLU);
    hipEventCreate(&stopLU);
    for (int i = 0; i < N; ++i) {
        hipEventRecord(startLU);
        scaleIndex<<<1,N>>>(d_U, d_L, N, i);
        hipEventRecord(stopLU);
        hipEventSynchronize(stopLU);
        float l1_time = 0;
        hipEventElapsedTime(&l1_time,startLU, stopLU);
        l_time += std::chrono::duration<double>(l1_time/1000);

        hipDeviceSynchronize();
        hipEventRecord(startLU);
        elimination<<<gridConfig, blockConfig>>>(d_L, d_U, N, i, TILE);
        hipEventRecord(stopLU);
        hipEventSynchronize(stopLU);
        float u1_time = 0;
        hipEventElapsedTime(&u1_time,startLU, stopLU);
        u_time += std::chrono::duration<double>(u1_time/1000);
    }

    hipMemcpy(L, d_L, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    auto start_sub = std::chrono::high_resolution_clock::now();
    forwardSubstitution(L, B, Y, N);
    backwardSubstitution(U, Y, X, N);
    auto end_sub = std::chrono::high_resolution_clock::now();

    lu_decomposition_time = l_time + u_time;
    total_time = l_time + u_time + end_sub - start_sub;

    std::cout << "Read time: " << read_time.count() << "s" << std::endl;
    std::cout << "L time: " << l_time.count() << "s" << std::endl;
    std::cout << "U time: " << u_time.count() << "s" << std::endl;
    std::cout << "LU decomposition time: " << lu_decomposition_time.count() << "s" << std::endl;
    std::cout << "Total time: " << total_time.count() << "s" << std::endl;

    // write to file
    writeToFile(argv[2], N, L, U, X);

    // Free device memory
    hipFree(d_L);
    hipFree(d_U);
    // Free host memory
    free(A);
    free(B);
    free(X);
    return 0;
}
