#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <chrono>

std::chrono::duration<double> read_time(0);
std::chrono::duration<double> l_time(0);
std::chrono::duration<double> u_time(0);
std::chrono::duration<double> lu_decomposition_time(0);
std::chrono::duration<double> total_time(0);

#define TILE 16

void readInput(const char *filename, int &N, double **A, double **B) {
    std::ifstream infile(filename);
    if (!infile) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    infile >> N;

    *A = (double *)malloc(N * N * sizeof(double));
    *B = (double *)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            infile >> (*A)[i * N + j];

    for (int i = 0; i < N; ++i)
        infile >> (*B)[i];

    infile.close();
}

// Row elimination Kernel
__global__ void elimination(double *A, double* L, double* U, int n, int index, int bsize) {
    int idThread = threadIdx.x;
    int idBlock = blockIdx.x;

    int pivotRow = (index * n);
    int currentRow = (((bsize * idBlock) + idThread) * n);
    int start = currentRow + index;
    int end = currentRow + n;

    if (currentRow > pivotRow && currentRow < n * n) {
        for (int i = currentRow; i < start + 1; i++) {
            U[i] = 0;
        }
        for (int i = start + 1; i < end; ++i) {
            U[i] = U[i] - (L[start] * U[pivotRow + (i - currentRow)]);
        }
    }
}

__global__ void scaleIndex(double* U, double *L, int n, int index) {
    int start = (index * n + index);
    int end = (index * n + n);
    L[start] = 1;
    for (int i = index + 1; i < n; ++i) {
        L[i * n + index] = (U[i * n + index] / U[start]);
    }

}

void forwardSubstitution(double* L, double* B, double* Y, int N) {
    for (int row = 0; row < N; row++) {
        Y[row] = B[row];
        for (int j = 0; j < row; j++) {
            Y[row] -= L[row * N + j] * Y[j];
        }
    }
}

void backwardSubstitution(double* U, double* Y, double* X, int N) {
    for (int row = N - 1; row >= 0; row--) {
        X[row] = Y[row];
        for (int j = row + 1; j < N; j++) {
            X[row] -= U[row * N + j] * X[j];
        }
        X[row] /= U[row * N + row];
    }
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: ./cuda " << "<input_file> " << "<output_file>" << std::endl;
        exit(EXIT_FAILURE);
    }
    int N;
    double *A, *B, *X, *Y;
    auto read_start = std::chrono::high_resolution_clock::now();
    readInput(argv[1], N, &A, &B);
    auto read_end = std::chrono::high_resolution_clock::now();
    read_time = read_end - read_start;

    X = (double *)malloc(N * sizeof(double));
    Y = (double *)malloc(N * sizeof(double));
    double *d_A, *d_L, *d_U;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_L, N * N * sizeof(double));
    hipMalloc(&d_U, N * N * sizeof(double));
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_U, A, N * N * sizeof(double), hipMemcpyHostToDevice);

    // dim3 gridConfig((N/TILE) + ((N%TILE) ? 1 : 0), 1, 1);
    // dim3 blockConfig(TILE, 1, 1);
    int gridConfig = (N/TILE) + ((N%TILE) ? 1 : 0);
    int blockConfig = TILE;

    double* L = (double*)malloc(N * N * sizeof(double));
    double* U = (double*)malloc(N * N * sizeof(double));

    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < N; ++i) {
        scaleIndex<<<1,1>>>(d_U, d_L, N, i);
        elimination<<<gridConfig, TILE>>>(d_A, d_L, d_U, N, i, TILE);
    }

    hipMemcpy(A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L, d_L, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    auto end = std::chrono::high_resolution_clock::now();

    lu_decomposition_time = end - start;

    auto start_sub = std::chrono::high_resolution_clock::now();
    forwardSubstitution(L, B, Y, N);
    backwardSubstitution(U, Y, X, N);
    auto end_sub = std::chrono::high_resolution_clock::now();

    total_time =  lu_decomposition_time + end_sub - start_sub;

    std::cout << "Read time: " << read_time.count() << "s" << std::endl;
    std::cout << "LU decomposition time: " << lu_decomposition_time.count() << "s" << std::endl;
    std::cout << "Total time: " << total_time.count() << "s" << std::endl;

    std::ofstream outfile(argv[2]);
    if (!outfile) {
        std::cerr << "Error opening file for writing: "<< argv[2] << std::endl;
        exit(EXIT_FAILURE);
    }

    outfile << N << std::endl;

    // Write L matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << L[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write U matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << U[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write solution vector X
    for (int i = 0; i < N; i++) {
        outfile << X[i] << std::endl;
    }

    outfile.close();

    // Free device memory
    hipFree(d_A);

    // Free host memory
    free(A);
    free(B);
    free(X);
    return 0;
}
