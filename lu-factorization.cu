#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

void readInput(const char *filename, int &N, double **A, double **B) {
    std::ifstream infile(filename);
    if (!infile) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    infile >> N;

    *A = (double *)malloc(N * N * sizeof(double));
    *B = (double *)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            infile >> (*A)[i * N + j];

    for (int i = 0; i < N; ++i)
        infile >> (*B)[i];

    infile.close();
}

__global__ void luDecomposition(double *A, double *L, double *U, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N) {
        for (int j = 0; j < N; j++) {
            if (j < row) {
                L[row * N + j] = A[row * N + j]; // L below diagonal
                U[row * N + j] = A[row * N + j]; // U has zeros below diagonal
            } else {
                U[row * N + j] = A[row * N + j]; // U above diagonal
                L[row * N + j] = (row == j) ? 1.0 : 0.0; // L diagonal elements
            }
        }
    }

    __syncthreads(); // Ensure all threads have updated L and U

    // Perform elimination
    for (int k = 0; k < N; k++) {
        if (row > k) {
            double factor = U[k * N + k] != 0 ? (U[row * N + k] / U[k * N + k]) : 0.0;
            for (int j = k + 1; j < N; j++) {
                U[row * N + j] -= factor * U[k * N + j];
            }
            U[row * N + k] = 0.0;
            L[row * N + k] = factor;
        }
    }
}


void forwardSubstitution(double* A, double* B, double* Y, int N) {
    for (int row = 0; row < N; row++) {
        Y[row] = B[row];
        for (int j = 0; j < row; j++) {
            Y[row] -= A[row * N + j] * Y[j];
        }
    }
}

void backwardSubstitution(double* A, double* B, double* X, int N) {
    for (int row = N - 1; row >= 0; row--) {
        X[row] = B[row];
        for (int j = row + 1; j < N; j++) {
            X[row] -= A[row * N + j] * X[j];
        }
        X[row] /= A[row * N + row];
    }
}

int main() {
    int N;
    double *A, *B, *X;
    readInput("input10.txt", N, &A, &B);
    X = (double *)malloc(N * sizeof(double));

    // print N, A, B
    printf("N: %d\n", N);
    printf("A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", A[i * N + j]);
        }
        printf("\n");
    }
    printf("B:\n");
    for (int i = 0; i < N; i++) {
        printf("%f\n", B[i]);
    }

    double *d_A, *d_B, *d_L, *d_U, *d_Y, *d_X;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_B, N * sizeof(double));
    hipMalloc(&d_L, N * N * sizeof(double));
    hipMalloc(&d_U, N * N * sizeof(double));
    hipMalloc(&d_Y, N * sizeof(double));
    hipMalloc(&d_X, N * sizeof(double));

    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridConfig(1, 1, 1);
    dim3 blockConfig(1, N, 1);

    int sharedMemSize = N * N * sizeof(double) + N * sizeof(double) + N * sizeof(double);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    luDecomposition<<<gridConfig, blockConfig>>>(d_A, d_L, d_U, N);


    double* L = (double*)malloc(N * N * sizeof(double));
    double* U = (double*)malloc(N * N * sizeof(double));

    hipMemcpy(L, d_L, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);

    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("LU Decomposition time: %f ms\n", milliseconds);

    // print L and U
    printf("L:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", L[i * N + j]);
        }
        printf("\n");
    }
    printf("U:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", U[i * N + j]);
        }
        printf("\n");
    }
    
    forwardSubstitution(L, B, X, N);
    backwardSubstitution(U, X, X, N);

    printf("X:\n");
    for (int i = 0; i < N; i++) {
        printf("%f\n", X[i]);
    }

    std::ofstream outfile("output.txt");
    if (!outfile) {
        std::cerr << "Error opening file for writing: output.txt" << std::endl;
        exit(EXIT_FAILURE);
    }

    outfile << N << std::endl;

    // Write L matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << L[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write U matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            outfile << U[i * N + j] << " ";
        }
        outfile << std::endl;
    }

    // Write solution vector X
    for (int i = 0; i < N; i++) {
        outfile << X[i] << std::endl;
    }

    outfile.close();


    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_L);
    hipFree(d_U);
    hipFree(d_Y);
    hipFree(d_X);

    // Free host memory
    free(A);
    free(B);
    free(X);
    return 0;
}